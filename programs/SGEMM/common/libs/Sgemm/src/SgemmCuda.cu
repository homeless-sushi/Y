#include "hip/hip_runtime.h"
#include "Sgemm/Matrix.h"
#include "Sgemm/Sgemm.h"
#include "Sgemm/SgemmCuda.h"

#include <sstream>
#include <stdexcept>

#include <cmath>

#include "CudaError/CudaError.h"

namespace Sgemm
{
    __global__
    void kernel(
        float alpha,
        float beta,
        float* a,
        float* b,
        float* c,
        unsigned int m,
        unsigned int n,
        unsigned int p,
        unsigned int tileSize
    )
    {
        extern __shared__ float bCached[];

        unsigned int xCAbsIdx = blockIdx.x*tileSize + threadIdx.x;
        unsigned int yCAbsIdx = blockIdx.y*tileSize + threadIdx.y;

        float threadRes = 0;
        for(unsigned int tileIdx = 0; tileIdx < n/tileSize; ++tileIdx){

            __syncthreads();
            unsigned int xBAbsIdx = tileIdx*tileSize + threadIdx.x;
            unsigned int yBAbsIdx = yCAbsIdx;
            bCached[threadIdx.x*tileSize+threadIdx.y] = b[xBAbsIdx*p+yBAbsIdx];
            __syncthreads();

            for(unsigned int i=0; i<tileSize; ++i){
                unsigned int iOffset = 
                    (threadIdx.y + i < tileSize) ?
                    threadIdx.y + i :
                    threadIdx.y + i - tileSize;

                unsigned int xAAbsIdx = xCAbsIdx;
                unsigned int yAAbsIdx = tileIdx*tileSize + iOffset;

                threadRes += 
                    alpha * 
                    a[xAAbsIdx*n + yAAbsIdx] *
                    bCached[iOffset*tileSize + threadIdx.y];
            }
        }

        unsigned int linearizedCAbsIdx = xCAbsIdx*p + yCAbsIdx;
        c[linearizedCAbsIdx] = threadRes + beta * c[linearizedCAbsIdx];
    }

    SgemmCuda::SgemmCuda(
        float alpha,
        float beta,
        Matrix& a,
        Matrix& b,
        Matrix& c,
        unsigned int tileSize
    ):
        Sgemm(alpha, beta, a, b, c),
        tileSize_(tileSize)
    {

        if(
            a_.nrows()%tileSize_ != 0 ||
            b_.ncols()%tileSize_ != 0 ||
            c_.ncols()%tileSize_ != 0
        ){
            std::ostringstream errorMsg;
            errorMsg << "Matrix dimensions are not multiple of tile size:" << "\n"
                << "\tA is " << a.nrows() << "x"<< a.ncols() << "\n"
                << "\tB is " << b.nrows() << "x"<< b.ncols() << "\n"
                << "\tC is " << c.nrows() << "x"<< c.ncols() << "\n";
            throw std::runtime_error(errorMsg.str());
        }
        
        CudaErrorCheck(hipMalloc(&aDevice_, sizeof(float)*a_.nrows()*a_.ncols()));
        CudaErrorCheck(hipMalloc(&bDevice_, sizeof(float)*b_.nrows()*b_.ncols()));
        CudaErrorCheck(hipMalloc(&cDevice_, sizeof(float)*c_.nrows()*c_.ncols()));
#ifdef TIMERS
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
#endif //TIMERS
        CudaErrorCheck(hipMemcpy(
            aDevice_,
            a_.data(),
            sizeof(float)*a_.nrows()*a_.ncols(),
            hipMemcpyKind::hipMemcpyHostToDevice
        ));
        CudaErrorCheck(hipMemcpy(
            bDevice_,
            b_.data(),
            sizeof(float)*b_.nrows()*b_.ncols(),
            hipMemcpyKind::hipMemcpyHostToDevice
        ));
        CudaErrorCheck(hipMemcpy(
            cDevice_,
            c_.data(),
            sizeof(float)*c_.nrows()*c_.ncols(),
            hipMemcpyKind::hipMemcpyHostToDevice
        ));
#ifdef TIMERS
        hipEventRecord(stop);
        hipDeviceSynchronize();
        hipEventElapsedTime(&dataUploadTime, start, stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);
#endif //TIMERS
    }

    SgemmCuda::~SgemmCuda()
    {
        CudaErrorCheck(hipFree(aDevice_));
        CudaErrorCheck(hipFree(bDevice_));
        CudaErrorCheck(hipFree(cDevice_));
        aDevice_ = nullptr;
        bDevice_ = nullptr;
        cDevice_ = nullptr;
    }
    
    void SgemmCuda::run(){
        dim3 grid(c_.nrows()/tileSize_, c_.ncols()/tileSize_);
        dim3 block(tileSize_,tileSize_);
#ifdef TIMERS
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
#endif //TIMERS
        kernel<<<grid, block, sizeof(float)*tileSize_*tileSize_>>>(
            alpha_,
            beta_,
            aDevice_,
            bDevice_,
            cDevice_,
            a_.nrows(), 
            b_.nrows(),
            c_.nrows(),
            tileSize_
        );
#ifdef TIMERS
        CudaKernelErrorCheck();
        hipEventRecord(stop);
        hipDeviceSynchronize();
        hipEventElapsedTime(&kernelTime, start, stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);
#else
        hipDeviceSynchronize();
#endif //TIMERS
    }

    Matrix SgemmCuda::getResult(){
        float* resData_ = (float*) malloc(sizeof(float)*c_.nrows()*c_.ncols());
#ifdef TIMERS
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
#endif //TIMERS
        CudaErrorCheck(hipMemcpy(
            resData_,
            cDevice_,
            sizeof(float)*c_.nrows()*c_.ncols(),
            hipMemcpyKind::hipMemcpyDeviceToHost
        ));
#ifdef TIMERS
        hipEventRecord(stop);
        hipDeviceSynchronize();
        hipEventElapsedTime(&dataDownloadTime, start, stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);
#endif //TIMERS
        Matrix res(c_.nrows(), c_.ncols(), resData_);
        return res;
    }
}