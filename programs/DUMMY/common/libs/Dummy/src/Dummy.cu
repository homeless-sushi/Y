#include "hip/hip_runtime.h"
#include <Dummy/Dummy.h>

#include <vector>

#include <hip/hip_runtime.h>

#include "CudaError/CudaError.h"

namespace Dummy 
{
    
    Dummy::Dummy(
        std::vector<float> data,
        unsigned int gridLen,
        unsigned int blockLen,
        unsigned int times
    ) : 
        data{data},
        gridLen{gridLen},
        blockLen{blockLen},
        times{times}
    {
        CudaErrorCheck(hipMalloc(&gpu_in, sizeof(float)*data.size()));
        CudaErrorCheck(hipMalloc(&gpu_out, sizeof(float)*data.size()));
#ifdef TIMERS
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
#endif //TIMERS
        hipMemcpy(
            gpu_in,
            data.data(),
            sizeof(float)*data.size(),
            hipMemcpyHostToDevice
        );
        hipMemset(gpu_out, 0, sizeof(float)*data.size());
#ifdef TIMERS
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&dataUploadTime, start, stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);
#endif //TIMERS
    };

    Dummy::~Dummy()
    {
        CudaErrorCheck(hipFree(gpu_in));
        CudaErrorCheck(hipFree(gpu_out));
    };

    std::vector<float> Dummy::getResult()
    {
#ifdef TIMERS
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
#endif //TIMERS
        hipMemcpy(
            data.data(),
            gpu_out,
            sizeof(float)*data.size(),
            hipMemcpyDeviceToHost
        );
#ifdef TIMERS
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&dataDownloadTime, start, stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);
#endif //TIMERS
        return data;
    }

    __global__
    void dummyKernel(float* in, float* out, unsigned int n, unsigned int times)
    {
        unsigned int absolute_idx = blockDim.x * blockIdx.x + threadIdx.x;
        unsigned int stride = gridDim.x * blockDim.x;

        for(unsigned int t = 0; t < times; t++){
            for(int i = absolute_idx; i < n; i+=stride){
                out[i]=in[i] * i;
            }
            for(int i = absolute_idx; i < n; i+=stride){
                out[i]=in[i] / i;
            }
        }
    };

    __global__
    void dummyInfinteKernel(float* in, float* out, unsigned int n)
    {
        unsigned int absolute_idx = blockDim.x * blockIdx.x + threadIdx.x;
        unsigned int stride = gridDim.x * blockDim.x;

        while(true){
            for(int i = absolute_idx; i < n; i+=stride){
                out[i]=in[i] * i;
            }
            for(int i = absolute_idx; i < n; i+=stride){
                out[i]=in[i] / i;
            }
        }
    };

    void Dummy::run()
    {
#ifdef TIMERS
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
#endif //TIMERS
        dummyKernel<<<gridLen, blockLen>>>(gpu_in, gpu_out, data.size(), times);
#ifdef TIMERS
        hipEventRecord(stop);
        hipDeviceSynchronize();
        hipEventElapsedTime(&kernelTime, start, stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);
#else
        hipDeviceSynchronize();
#endif //TIMERS
    };

    void Dummy::runInfinite()
    {
#ifdef TIMERS
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
#endif //TIMERS
        dummyInfinteKernel<<<gridLen, blockLen>>>(gpu_in, gpu_out, data.size());
#ifdef TIMERS
        hipEventRecord(stop);
        hipDeviceSynchronize();
        hipEventElapsedTime(&kernelTime, start, stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);
#else
        hipDeviceSynchronize();
#endif //TIMERS
    };
}
