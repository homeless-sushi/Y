#include "hip/hip_runtime.h"
#include "Bfs/Bfs.h"
#include "Bfs/BfsCuda.h"

#include <vector>

#include <hip/hip_runtime.h>

#include "Graph/Graph.h"

namespace Bfs
{
    namespace
    {
        void createTextureObject(unsigned int* src, hipTextureObject_t* dst, size_t size)
        {
            hipResourceDesc resourceDesc;
            memset(&resourceDesc, 0, sizeof(resourceDesc));
            resourceDesc.resType = hipResourceTypeLinear;
            resourceDesc.res.linear.devPtr = src;
            resourceDesc.res.linear.desc.f = hipChannelFormatKindUnsigned;
            resourceDesc.res.linear.desc.x = sizeof(unsigned int)*CHAR_BIT;
            resourceDesc.res.linear.sizeInBytes = sizeof(unsigned int)*size;
            hipTextureDesc texDesc;
            memset(&texDesc, 0, sizeof(texDesc));
            texDesc.readMode = hipReadModeElementType;
            hipCreateTextureObject(dst, &resourceDesc, &texDesc, NULL);
        }

        __global__
        void kernel(
            unsigned int nVertices,
            unsigned int chunkSize,
            unsigned int *edgeOffsetsDevice,
            unsigned int *edgesDevice,
            hipTextureObject_t edgeOffsetsTexture,
            hipTextureObject_t edgesTexture,
            bool textureMemForEdgesOffsets,
            bool textureMemForEdges,
            int *costs,
            int currCost,
            bool *done)
        {
            const unsigned int startNode = blockIdx.x*chunkSize;
            const unsigned int stopNode = min(startNode + chunkSize, nVertices);
            for(unsigned int fromNode = startNode; fromNode < stopNode; fromNode++) {
                if(costs[fromNode] == currCost) {
                    unsigned int nodeEdgesStart;
                    unsigned int nodeEdgesEnd;
                    if(textureMemForEdgesOffsets){
                        nodeEdgesStart = tex1Dfetch<unsigned int>(edgeOffsetsTexture, fromNode);
                        nodeEdgesEnd = tex1Dfetch<unsigned int>(edgeOffsetsTexture, fromNode+1);
                    }else{
                        nodeEdgesStart = edgeOffsetsDevice[fromNode];
                        nodeEdgesEnd = edgeOffsetsDevice[fromNode+1];
                    }
                    
                    for(unsigned int i = nodeEdgesStart + threadIdx.x; i < nodeEdgesEnd; i+=blockDim.x) {
                        unsigned int toNode;

                        if(textureMemForEdges){
                            toNode = tex1Dfetch<unsigned int>(edgesTexture, i);
                        }else{
                            toNode = edgesDevice[i];
                        }

                        if(costs[toNode] == -1) {
                            costs[toNode] = currCost + 1;
                            *done = false;
                        }
                    }
                }
            }
        }
    }

    BfsCuda::BfsCuda(    
        Graph::Graph& graph,
        unsigned int source,
        unsigned int blockSize,
        unsigned int chunkFactor, 
        bool textureMemForEdgesOffsets,
        bool textureMemForEdges
    ) :
        Bfs(graph, source),
        blockSize_(blockSize),
        chunkFactor_(chunkFactor),
        textureMemForEdgesOffsets_(textureMemForEdgesOffsets),
        textureMemForEdges_(textureMemForEdges)
    {
        hipMalloc(&edgeOffsetsDevice_, sizeof(unsigned int)*graph.edgeOffsets.size());
        hipMemcpy(edgeOffsetsDevice_, graph.edgeOffsets.data(), sizeof(unsigned int)*graph.edgeOffsets.size(), hipMemcpyKind::hipMemcpyHostToDevice);
        if(textureMemForEdgesOffsets_){
            memset(&edgeOffsetsTexture_, 0, sizeof(hipTextureObject_t));
            createTextureObject(edgeOffsetsDevice_, &edgeOffsetsTexture_, graph.edgeOffsets.size());
        }

        hipMalloc(&edgesDevice_, sizeof(unsigned int)*graph.edges.size());
        hipMemcpy(edgesDevice_, graph.edges.data(), sizeof(unsigned int)*graph.edges.size(), hipMemcpyKind::hipMemcpyHostToDevice);
        if(textureMemForEdges_){
            memset(&edgesTexture_, 0, sizeof(hipTextureObject_t));
            createTextureObject(edgesDevice_, &edgesTexture_, graph.edges.size());
        }

        hipMalloc(&costsDevice_, sizeof(int)*graph.nVertices);
        hipMemset(costsDevice_, -1, sizeof(unsigned int)*graph.nVertices);
        hipMemset(costsDevice_ + source, 0, sizeof(unsigned int));

        hipMalloc(&doneDevice_, sizeof(bool));
        hipMemset(doneDevice_, true, sizeof(bool));
    }

    BfsCuda::~BfsCuda() 
    {
        hipFree(edgeOffsetsDevice_);
        hipFree(edgesDevice_);

        if(textureMemForEdgesOffsets_){
            hipDestroyTextureObject(edgeOffsetsTexture_);
        }
        if(textureMemForEdges_){
            hipDestroyTextureObject(edgesTexture_);
        }

        hipFree(costsDevice_);
        hipFree(doneDevice_);
    }

    bool BfsCuda::run()
    {
        hipMemset(doneDevice_, true, sizeof(bool));
            

        const unsigned int blockSize = blockSize_;
        const unsigned int chunkSize = blockSize * chunkFactor_;
        const unsigned int gridSize = (graph.nVertices + blockSize - 1)/chunkSize + 1;

        kernel<<<gridSize, blockSize>>>(
            graph.nVertices, 
            chunkSize,
            edgeOffsetsDevice_, 
            edgesDevice_,
            edgeOffsetsTexture_,
            edgesTexture_,
            textureMemForEdgesOffsets_,
            textureMemForEdges_,
            costsDevice_,
            currentCost,
            doneDevice_
        );
        
        currentCost++;
        bool done;
        hipMemcpy(&done, doneDevice_, sizeof(bool), hipMemcpyKind::hipMemcpyDeviceToHost);
        return done;
    }

    const std::vector<int>& BfsCuda::getResult() 
    {
        costsHost_.reserve(graph.nVertices);
        costsHost_.resize(costsHost_.capacity());
        hipMemcpy(costsHost_.data(), costsDevice_, sizeof(int)*graph.nVertices, hipMemcpyKind::hipMemcpyDeviceToHost);
        return costsHost_;
    };
}