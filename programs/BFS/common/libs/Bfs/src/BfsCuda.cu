#include "hip/hip_runtime.h"
#include "Bfs/Bfs.h"
#include "Bfs/BfsCuda.h"

#include <vector>

#include <hip/hip_runtime.h>

#include "Graph/Graph.h"

#include "CudaError/CudaError.h"

namespace Bfs
{
    namespace
    {
        void createTextureObject(unsigned int* src, hipTextureObject_t* dst, size_t size)
        {
            hipResourceDesc resourceDesc;
            memset(&resourceDesc, 0, sizeof(resourceDesc));
            resourceDesc.resType = hipResourceTypeLinear;
            resourceDesc.res.linear.devPtr = src;
            resourceDesc.res.linear.desc.f = hipChannelFormatKindUnsigned;
            resourceDesc.res.linear.desc.x = sizeof(unsigned int)*CHAR_BIT;
            resourceDesc.res.linear.sizeInBytes = sizeof(unsigned int)*size;
            hipTextureDesc texDesc;
            memset(&texDesc, 0, sizeof(texDesc));
            texDesc.readMode = hipReadModeElementType;
            CudaErrorCheck(
                hipCreateTextureObject(dst, &resourceDesc, &texDesc, NULL)
            );
        }

        __global__
        void kernel(
            unsigned int nVertices,
            unsigned int chunkSize,
            unsigned int *edgeOffsetsDevice,
            unsigned int *edgesDevice,
            hipTextureObject_t edgeOffsetsTexture,
            hipTextureObject_t edgesTexture,
            bool textureMemForEdgesOffsets,
            bool textureMemForEdges,
            int *costs,
            int currCost,
            bool *done)
        {
            const unsigned int startNode = blockIdx.x*chunkSize;
            const unsigned int stopNode = min(startNode + chunkSize, nVertices);
            for(unsigned int fromNode = startNode; fromNode < stopNode; fromNode++) {
                if(costs[fromNode] == currCost) {
                    unsigned int nodeEdgesStart;
                    unsigned int nodeEdgesEnd;
                    if(textureMemForEdgesOffsets){
                        nodeEdgesStart = tex1Dfetch<unsigned int>(edgeOffsetsTexture, fromNode);
                        nodeEdgesEnd = tex1Dfetch<unsigned int>(edgeOffsetsTexture, fromNode+1);
                    }else{
                        nodeEdgesStart = edgeOffsetsDevice[fromNode];
                        nodeEdgesEnd = edgeOffsetsDevice[fromNode+1];
                    }
                    
                    for(unsigned int i = nodeEdgesStart + threadIdx.x; i < nodeEdgesEnd; i+=blockDim.x) {
                        unsigned int toNode;

                        if(textureMemForEdges){
                            toNode = tex1Dfetch<unsigned int>(edgesTexture, i);
                        }else{
                            toNode = edgesDevice[i];
                        }

                        if(costs[toNode] == -1) {
                            costs[toNode] = currCost + 1;
                            *done = false;
                        }
                    }
                }
            }
        }
    }

    BfsCuda::BfsCuda(    
        Graph::Graph& graph,
        unsigned int source,
        unsigned int blockSize,
        unsigned int chunkFactor, 
        bool textureMemForEdgesOffsets,
        bool textureMemForEdges
    ) :
        Bfs(graph, source),
        blockSize_(blockSize),
        chunkFactor_(chunkFactor),
        textureMemForEdgesOffsets_(textureMemForEdgesOffsets),
        textureMemForEdges_(textureMemForEdges)
    {

        CudaErrorCheck(hipMalloc(
            &edgeOffsetsDevice_,
            sizeof(unsigned int)*graph.edgeOffsets.size()
        ));
        CudaErrorCheck(hipMemcpy(
                edgeOffsetsDevice_,
                graph.edgeOffsets.data(),
                sizeof(unsigned int)*graph.edgeOffsets.size(), 
                hipMemcpyKind::hipMemcpyHostToDevice
        ));
        if(textureMemForEdgesOffsets_){
            memset(&edgeOffsetsTexture_, 0, sizeof(hipTextureObject_t));
            createTextureObject(
                    edgeOffsetsDevice_,
                    &edgeOffsetsTexture_,
                    graph.edgeOffsets.size()
            );
        }

        CudaErrorCheck(
            hipMalloc(&edgesDevice_, sizeof(unsigned int)*graph.edges.size())
        );
        CudaErrorCheck(hipMemcpy(
            edgesDevice_,
            graph.edges.data(),
            sizeof(unsigned int)*graph.edges.size(),
            hipMemcpyKind::hipMemcpyHostToDevice
        ));
        if(textureMemForEdges_){
            memset(&edgesTexture_, 0, sizeof(hipTextureObject_t));
            createTextureObject(
                edgesDevice_,
                &edgesTexture_,
                graph.edges.size()
            );
        }

        CudaErrorCheck(
            hipMalloc(&costsDevice_, sizeof(int)*graph.nVertices)
        );
        CudaErrorCheck(
            hipMemset(costsDevice_, -1, sizeof(unsigned int)*graph.nVertices)
        );
        CudaErrorCheck(
            hipMemset(costsDevice_ + source, 0, sizeof(unsigned int))
        );

        CudaErrorCheck(
            hipMalloc(&doneDevice_, sizeof(bool))
        );
        CudaErrorCheck(
            hipMemset(doneDevice_, true, sizeof(bool))
        );
    }

    BfsCuda::~BfsCuda() 
    {   
        CudaErrorCheck(hipFree(edgeOffsetsDevice_));
        CudaErrorCheck(hipFree(edgesDevice_));

        if(textureMemForEdgesOffsets_){
            CudaErrorCheck(hipDestroyTextureObject(edgeOffsetsTexture_));
        }
        if(textureMemForEdges_){
            CudaErrorCheck(hipDestroyTextureObject(edgesTexture_));
        }

        CudaErrorCheck(hipFree(costsDevice_));
        CudaErrorCheck(hipFree(doneDevice_));
    }

    bool BfsCuda::run()
    {
        CudaErrorCheck(hipMemset(doneDevice_, true, sizeof(bool)));

        const unsigned int blockSize = blockSize_;
        const unsigned int chunkSize = blockSize * chunkFactor_;
        const unsigned int gridSize = (graph.nVertices + blockSize - 1)/chunkSize + 1;

        kernel<<<gridSize, blockSize>>>(
            graph.nVertices, 
            chunkSize,
            edgeOffsetsDevice_, 
            edgesDevice_,
            edgeOffsetsTexture_,
            edgesTexture_,
            textureMemForEdgesOffsets_,
            textureMemForEdges_,
            costsDevice_,
            currentCost,
            doneDevice_
        );
        CudaKernelErrorCheck();
        
        currentCost++;
        bool done;
        CudaErrorCheck(hipMemcpy(
                &done,
                doneDevice_,
                sizeof(bool),
                hipMemcpyKind::hipMemcpyDeviceToHost
        ));
        return done;
    }

    const std::vector<int>& BfsCuda::getResult() 
    {
        costsHost_.reserve(graph.nVertices);
        costsHost_.resize(costsHost_.capacity());
        CudaErrorCheck(hipMemcpy(
            costsHost_.data(),
            costsDevice_,
            sizeof(int)*graph.nVertices,
            hipMemcpyKind::hipMemcpyDeviceToHost
        ));
        return costsHost_;
    };
}