#include "hip/hip_runtime.h"
#include "Bfs/Bfs.h"
#include "Bfs/BfsCuda.h"

#include <vector>

#include <hip/hip_runtime.h>

#include "Graph/Graph.h"

#include "CudaError/CudaError.h"

namespace Bfs
{
    namespace
    {
        void createTextureObject(unsigned int* src, hipTextureObject_t* dst, size_t size)
        {
            hipResourceDesc resourceDesc;
            memset(&resourceDesc, 0, sizeof(resourceDesc));
            resourceDesc.resType = hipResourceTypeLinear;
            resourceDesc.res.linear.devPtr = src;
            resourceDesc.res.linear.desc.f = hipChannelFormatKindUnsigned;
            resourceDesc.res.linear.desc.x = sizeof(unsigned int)*CHAR_BIT;
            resourceDesc.res.linear.sizeInBytes = sizeof(unsigned int)*size;
            hipTextureDesc texDesc;
            memset(&texDesc, 0, sizeof(texDesc));
            texDesc.readMode = hipReadModeElementType;
            CudaErrorCheck(
                hipCreateTextureObject(dst, &resourceDesc, &texDesc, NULL)
            );
        }

        __global__
        void kernel(
            unsigned int nVertices,
            unsigned int chunkSize,
            unsigned int *edgeOffsetsDevice,
            unsigned int *edgesDevice,
            hipTextureObject_t edgeOffsetsTexture,
            hipTextureObject_t edgesTexture,
            bool textureMemForEdgesOffsets,
            bool textureMemForEdges,
            int *costs,
            int currCost,
            bool *done)
        {
            const unsigned int startNode = blockIdx.x*chunkSize;
            const unsigned int stopNode = min(startNode + chunkSize, nVertices);
            for(unsigned int fromNode = startNode; fromNode < stopNode; fromNode++) {
                if(costs[fromNode] == currCost) {
                    unsigned int nodeEdgesStart;
                    unsigned int nodeEdgesEnd;
                    if(textureMemForEdgesOffsets){
                        nodeEdgesStart = tex1Dfetch<unsigned int>(edgeOffsetsTexture, fromNode);
                        nodeEdgesEnd = tex1Dfetch<unsigned int>(edgeOffsetsTexture, fromNode+1);
                    }else{
                        nodeEdgesStart = edgeOffsetsDevice[fromNode];
                        nodeEdgesEnd = edgeOffsetsDevice[fromNode+1];
                    }
                    
                    for(unsigned int i = nodeEdgesStart + threadIdx.x; i < nodeEdgesEnd; i+=blockDim.x) {
                        unsigned int toNode;

                        if(textureMemForEdges){
                            toNode = tex1Dfetch<unsigned int>(edgesTexture, i);
                        }else{
                            toNode = edgesDevice[i];
                        }

                        if(costs[toNode] == -1) {
                            costs[toNode] = currCost + 1;
                            *done = false;
                        }
                    }
                }
            }
        }
    }

    BfsCuda::BfsCuda(    
        Graph::Graph& graph,
        unsigned int source,
        unsigned int blockSize,
        unsigned int chunkFactor, 
        bool textureMemForEdgesOffsets,
        bool textureMemForEdges
    ) :
        Bfs(graph, source),
        blockSize_(blockSize),
        chunkFactor_(chunkFactor),
        textureMemForEdgesOffsets_(textureMemForEdgesOffsets),
        textureMemForEdges_(textureMemForEdges)
    {

        CudaErrorCheck(
            hipMalloc(&edgeOffsetsDevice_,sizeof(unsigned int)*graph.edgeOffsets.size())
        );
        CudaErrorCheck(
            hipMalloc(&edgesDevice_, sizeof(unsigned int)*graph.edges.size())
        );
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
        hipMemcpy(
            edgeOffsetsDevice_,
            graph.edgeOffsets.data(),
            sizeof(unsigned int)*graph.edgeOffsets.size(), 
            hipMemcpyKind::hipMemcpyHostToDevice
        );
        hipMemcpy(
            edgesDevice_,
            graph.edges.data(),
            sizeof(unsigned int)*graph.edges.size(),
            hipMemcpyKind::hipMemcpyHostToDevice
        );
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&dataUploadTime, start, stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        if(textureMemForEdgesOffsets_){
            memset(&edgeOffsetsTexture_, 0, sizeof(hipTextureObject_t));
            createTextureObject(
                    edgeOffsetsDevice_,
                    &edgeOffsetsTexture_,
                    graph.edgeOffsets.size()
            );
        }
        if(textureMemForEdges_){
            memset(&edgesTexture_, 0, sizeof(hipTextureObject_t));
            createTextureObject(
                edgesDevice_,
                &edgesTexture_,
                graph.edges.size()
            );
        }

        CudaErrorCheck(hipMalloc(&costsDevice_, sizeof(int)*graph.nVertices));
        hipMemset(costsDevice_, -1, sizeof(unsigned int)*graph.nVertices);
        hipMemset(costsDevice_ + source, 0, sizeof(unsigned int));
        CudaErrorCheck(hipMalloc(&doneDevice_, sizeof(bool)));
        hipMemset(doneDevice_, true, sizeof(bool));
    }

    BfsCuda::~BfsCuda() 
    {   
        CudaErrorCheck(hipFree(edgeOffsetsDevice_));
        CudaErrorCheck(hipFree(edgesDevice_));

        if(textureMemForEdgesOffsets_){
            CudaErrorCheck(hipDestroyTextureObject(edgeOffsetsTexture_));
        }
        if(textureMemForEdges_){
            CudaErrorCheck(hipDestroyTextureObject(edgesTexture_));
        }

        CudaErrorCheck(hipFree(costsDevice_));
        CudaErrorCheck(hipFree(doneDevice_));
    }

    void BfsCuda::run()
    {
        const unsigned int blockSize = blockSize_;
        const unsigned int chunkSize = blockSize * chunkFactor_;
        const unsigned int gridSize = (graph.nVertices + blockSize - 1)/chunkSize + 1;

        bool done = false;
        while(!done){
            hipMemset(doneDevice_, true, sizeof(bool));
            hipEvent_t start, stop;
            hipEventCreate(&start);
            hipEventCreate(&stop);
            hipEventRecord(start);
            kernel<<<gridSize, blockSize>>>(
                graph.nVertices, 
                chunkSize,
                edgeOffsetsDevice_, 
                edgesDevice_,
                edgeOffsetsTexture_,
                edgesTexture_,
                textureMemForEdgesOffsets_,
                textureMemForEdges_,
                costsDevice_,
                currentCost,
                doneDevice_
            );
            hipEventRecord(stop);
            hipDeviceSynchronize();
            float kernelTime;
            hipEventElapsedTime(&kernelTime, start, stop);
            kernelTotalTime+=kernelTime;
            hipEventDestroy(start);
            hipEventDestroy(stop);
            hipMemcpy(
                &done,
                doneDevice_,
                sizeof(bool),
                hipMemcpyKind::hipMemcpyDeviceToHost
            );
            currentCost++;
        }
    }

    const std::vector<int>& BfsCuda::getResult() 
    {
        costsHost_.reserve(graph.nVertices);
        costsHost_.resize(costsHost_.capacity());
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);
        hipMemcpy(
            costsHost_.data(),
            costsDevice_,
            sizeof(int)*graph.nVertices,
            hipMemcpyKind::hipMemcpyDeviceToHost
        );
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&dataDownloadTime, start, stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        return costsHost_;
    };
}